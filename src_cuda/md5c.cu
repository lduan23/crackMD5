/* MD5C.cpp is the cpp version of MD5C.c 
** in order to utilize timing module in cpp, 
** as well as better programming flexibility
*/

#include <cstdlib>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define S11 7
#define S12 12
#define S13 17
#define S14 22
#define S21 5
#define S22 9
#define S23 14
#define S24 20
#define S31 4
#define S32 11
#define S33 16
#define S34 23
#define S41 6
#define S42 10
#define S43 15
#define S44 21

/* F, G, H and I are basic MD5 functions */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4.
Rotation is separate from addition to prevent recomputation. */
#define FF(a, b, c, d, x, s, ac) { \
    (a) += F ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
}

#define GG(a, b, c, d, x, s, ac) { \
    (a) += G ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
}

#define HH(a, b, c, d, x, s, ac) { \
    (a) += H ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
}

#define II(a, b, c, d, x, s, ac) { \
    (a) += I ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
}

__device__ inline void getHash(unsigned char *data, uint32_t length, uint32_t *a1, uint32_t *b1, uint32_t *c1, uint32_t *d1) {

    // init with magic constants
    const uint32_t a0 = 0x67452301;
    const uint32_t b0 = 0xEFCDAB89;
    const uint32_t c0 = 0x98BADCFE;
    const uint32_t d0 = 0x10325476;

    // Calculate the padding is kind of mystery - temporarily!
    uint32_t padding[14] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0};

    int i = 0;
    for (i = 0; i < length; i++) {
        padding[i / 4] |= data[i] << ((i % 4) * 8);
    }

    padding[i / 4] |= 0x80 << ((i % 4) * 8);

    uint32_t bitlen = length * 8;

    #define in0  (padding[0])
    #define in1  (padding[1])
    #define in2  (padding[2])
    #define in3  (padding[3])
    #define in4  (padding[4])
    #define in5  (padding[5])
    #define in6  (padding[6])
    #define in7  (padding[7])
    #define in8  (padding[8])
    #define in9  (padding[9])
    #define in10 (padding[10])
    #define in11 (padding[11])
    #define in12 (padding[12])
    #define in13 (padding[13])
    #define in14 (bitlen)
    #define in15 (0)

    uint32_t a = a0;
    uint32_t b = b0;
    uint32_t c = c0;
    uint32_t d = d0;
        
    /* Round 1 */
    FF ( a, b, c, d, in0 , S11, 3614090360); /* 1 */
    FF ( d, a, b, c, in1 , S12, 3905402710); /* 2 */
    FF ( c, d, a, b, in2 , S13,  606105819); /* 3 */
    FF ( b, c, d, a, in3 , S14, 3250441966); /* 4 */
    FF ( a, b, c, d, in4 , S11, 4118548399); /* 5 */
    FF ( d, a, b, c, in5 , S12, 1200080426); /* 6 */
    FF ( c, d, a, b, in6 , S13, 2821735955); /* 7 */
    FF ( b, c, d, a, in7 , S14, 4249261313); /* 8 */
    FF ( a, b, c, d, in8 , S11, 1770035416); /* 9 */
    FF ( d, a, b, c, in9 , S12, 2336552879); /* 10 */
    FF ( c, d, a, b, in10, S13, 4294925233); /* 11 */
    FF ( b, c, d, a, in11, S14, 2304563134); /* 12 */
    FF ( a, b, c, d, in12, S11, 1804603682); /* 13 */
    FF ( d, a, b, c, in13, S12, 4254626195); /* 14 */
    FF ( c, d, a, b, in14, S13, 2792965006); /* 15 */
    FF ( b, c, d, a, in15, S14, 1236535329); /* 16 */

    /* Round 2 */
    GG ( a, b, c, d, in1 , S21, 4129170786); /* 17 */
    GG ( d, a, b, c, in6 , S22, 3225465664); /* 18 */
    GG ( c, d, a, b, in11, S23,  643717713); /* 19 */
    GG ( b, c, d, a, in0 , S24, 3921069994); /* 20 */
    GG ( a, b, c, d, in5 , S21, 3593408605); /* 21 */
    GG ( d, a, b, c, in10, S22,   38016083); /* 22 */
    GG ( c, d, a, b, in15, S23, 3634488961); /* 23 */
    GG ( b, c, d, a, in4 , S24, 3889429448); /* 24 */
    GG ( a, b, c, d, in9 , S21,  568446438); /* 25 */
    GG ( d, a, b, c, in14, S22, 3275163606); /* 26 */
    GG ( c, d, a, b, in3 , S23, 4107603335); /* 27 */
    GG ( b, c, d, a, in8 , S24, 1163531501); /* 28 */
    GG ( a, b, c, d, in13, S21, 2850285829); /* 29 */
    GG ( d, a, b, c, in2 , S22, 4243563512); /* 30 */
    GG ( c, d, a, b, in7 , S23, 1735328473); /* 31 */
    GG ( b, c, d, a, in12, S24, 2368359562); /* 32 */

    /* Round 3 */
    HH ( a, b, c, d, in5 , S31, 4294588738); /* 33 */
    HH ( d, a, b, c, in8 , S32, 2272392833); /* 34 */
    HH ( c, d, a, b, in11, S33, 1839030562); /* 35 */
    HH ( b, c, d, a, in14, S34, 4259657740); /* 36 */
    HH ( a, b, c, d, in1 , S31, 2763975236); /* 37 */
    HH ( d, a, b, c, in4 , S32, 1272893353); /* 38 */
    HH ( c, d, a, b, in7 , S33, 4139469664); /* 39 */
    HH ( b, c, d, a, in10, S34, 3200236656); /* 40 */
    HH ( a, b, c, d, in13, S31,  681279174); /* 41 */
    HH ( d, a, b, c, in0 , S32, 3936430074); /* 42 */
    HH ( c, d, a, b, in3 , S33, 3572445317); /* 43 */
    HH ( b, c, d, a, in6 , S34,   76029189); /* 44 */
    HH ( a, b, c, d, in9 , S31, 3654602809); /* 45 */
    HH ( d, a, b, c, in12, S32, 3873151461); /* 46 */
    HH ( c, d, a, b, in15, S33,  530742520); /* 47 */
    HH ( b, c, d, a, in2 , S34, 3299628645); /* 48 */

    /* Round 4 */
    II ( a, b, c, d, in0 , S41, 4096336452); /* 49 */
    II ( d, a, b, c, in7 , S42, 1126891415); /* 50 */
    II ( c, d, a, b, in14, S43, 2878612391); /* 51 */
    II ( b, c, d, a, in5 , S44, 4237533241); /* 52 */
    II ( a, b, c, d, in12, S41, 1700485571); /* 53 */
    II ( d, a, b, c, in3 , S42, 2399980690); /* 54 */
    II ( c, d, a, b, in10, S43, 4293915773); /* 55 */
    II ( b, c, d, a, in1 , S44, 2240044497); /* 56 */
    II ( a, b, c, d, in8 , S41, 1873313359); /* 57 */
    II ( d, a, b, c, in15, S42, 4264355552); /* 58 */
    II ( c, d, a, b, in6 , S43, 2734768916); /* 59 */
    II ( b, c, d, a, in13, S44, 1309151649); /* 60 */
    II ( a, b, c, d, in4 , S41, 4149444226); /* 61 */
    II ( d, a, b, c, in11, S42, 3174756917); /* 62 */
    II ( c, d, a, b, in2 , S43,  718787259); /* 63 */
    II ( b, c, d, a, in9 , S44, 3951481745); /* 64 */

    a += a0;
    b += b0;
    c += c0;
    d += d0;

    *a1 = a;
    *b1 = b;
    *c1 = c;
    *d1 = d;
}